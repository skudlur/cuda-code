#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <stdio.h>

__global__ void Vecadd()
{
    printf("Hello from GPU \n");
}

int main()
{
    int N = 10;
    //Kernel invocation with N threads
    Vecadd << <1, N >> > ();

    return 0;
}
