#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_threadIds()
{
    printf("threadIdx.x : %d, threadIdx.y : %d, threadIdx.z : %d \n",
   	 threadIdx.x, threadIdx.y, threadIdx.z);
}

__global__ void print_block()
{
    printf("blockIdx.x : %d, blockIdx.y : %d, blockIdx.z : %d, blockDim.x : %d, blockDim.y : %d, blockDim.z : %d, gridDim.x : %d, gridDim.y : %d, gridDim.z, %d \n",
   	 blockIdx.x, blockIdx.y, blockIdx.z, blockDim.x, blockDim.y, blockDim.z, gridDim.x, gridDim.y, gridDim.z);
}

int main()
{
    int nx, ny;
    nx = 16;
    ny = 16;

    dim3 block(8, 8);
    dim3 grid(nx / block.x, ny / block.y);

    print_block << <grid, block >> > ();
    hipDeviceSynchronize(); //we wait till our kernel has finished its execution.
    hipDeviceReset();
    return 0;
}
